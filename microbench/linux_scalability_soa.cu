#include "hip/hip_runtime.h"
#define NDEBUG
#include <chrono>
#include <stdio.h>
#include <assert.h>
#include <inttypes.h>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}


#include "allocator/soa_allocator.h"

#define THREADS_PER_BLOCK 256
#define NUM_BLOCKS 1024

class DummyClass {
 public:
  static const uint8_t kTypeId = 0;
  static const int kObjectSize = 8;
  static const uint8_t kBlockSize = 64;
};

__device__ SoaAllocator<64*64*64*64, DummyClass> memory_allocator;

__global__ void  benchmark(int num_iterations, DummyClass** ptrs) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  DummyClass** my_ptrs = ptrs + tid*num_iterations;

  for (int i = 0; i < num_iterations; ++i) {
    my_ptrs[i] = memory_allocator.make_new<DummyClass>();
  }

  for (int i = 0; i < num_iterations; ++i) {
    memory_allocator.free(my_ptrs[i]);
  }
}

__device__ void initialize_allocator() {
  memory_allocator.initialize();
}

__global__ void init_memory_system() {
  initialize_allocator();
}

int main() {
  DummyClass** ptr_storage;
  hipMalloc((void**) &ptr_storage, sizeof(void*)*64*64*64*64);
  gpuErrchk(hipDeviceSynchronize());

  for (int i = 0; i < 1000; ++i) {
    init_memory_system<<<256, 512>>>();
    gpuErrchk(hipDeviceSynchronize());

    auto time_before = std::chrono::system_clock::now();
    benchmark<<<64, 256>>>(i, ptr_storage);
    gpuErrchk(hipDeviceSynchronize());
    auto time_after = std::chrono::system_clock::now();
    int time_running = std::chrono::duration_cast<std::chrono::microseconds>(
        time_after - time_before).count();
    printf("%i\n", time_running);
  }
}
