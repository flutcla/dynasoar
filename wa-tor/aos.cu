#include "hip/hip_runtime.h"
#include <stdio.h>

#include <SDL2/SDL2_gfxPrimitives.h>


#define SPAWN_THRESHOLD 5
#define ENERGY_BOOST 3
#define GRID_SIZE_X 400
#define GRID_SIZE_Y 300

__device__ uint32_t random_number(uint32_t& state, uint32_t max) {
  // Advance and return random state.
  // Source: https://en.wikipedia.org/wiki/Lehmer_random_number_generator
  state = static_cast<uint32_t>(
      static_cast<uint64_t>(state) * 279470273u) % 0xfffffffb;
  return state % max;
}

class Agent;
class Fish;
class Shark;

class Cell {
 private:
  // left, top, right, bottom
  Cell* neighbors_[4];

  Agent* agent_ = nullptr;

  uint32_t random_state_;

  // left, top, right, bottom, self
  bool neighbor_request_[5];

 public:
  __device__ Cell(uint32_t random_state) : random_state_(random_state) {
    for (int i = 0; i < 5; ++i) {
      neighbor_request_[i] = false;
    }
  }

  __device__ Agent* agent() const {
    return agent_;
  }

  __device__ void set_agent(Agent* agent) {
    agent_ = agent;
  }

  __device__ bool is_free() const {
    return agent_ == nullptr;
  }

  __device__ bool has_fish() const;

  __device__ bool has_shark() const;

  __device__ uint32_t& random_state() {
    return random_state_;
  }

  __device__ void set_neighbors(Cell* left, Cell* top,
                                Cell* right, Cell* bottom) {
    neighbors_[0] = left;
    neighbors_[1] = top;
    neighbors_[2] = right;
    neighbors_[3] = bottom;
  }

  template<bool(Cell::*predicate)() const>
  __device__ bool request_random_neighbor(uint32_t& random_state) {
    uint8_t candidates[4];
    uint8_t num_candidates = 0;

    for (int i = 0; i < 4; ++i) {
      if ((neighbors_[i]->*predicate)()) {
        candidates[num_candidates++] = i;
      }
    }

    if (num_candidates == 0) {
      return false;
    } else {
      uint32_t selected_index = random_number(random_state, num_candidates);
      uint8_t selected = candidates[selected_index];
      uint8_t neighbor_index = (selected + 2) % 4;
      neighbors_[selected]->neighbor_request_[neighbor_index] = true;
      return true;
    }
  }

  __device__ void request_random_free_neighbor(uint32_t& random_state) {
    if (!request_random_neighbor<&Cell::is_free>(random_state)) {
      neighbor_request_[4] = true;
    }
  }

  __device__ void request_random_fish_neighbor(uint32_t& random_state) {
    if (!request_random_neighbor<&Cell::has_fish>(random_state)) {
      // No fish found. Look for free cell.
      if (!request_random_neighbor<&Cell::is_free>(random_state)) {
        neighbor_request_[4] = true;
      }
    }
  }

  __device__ void kill();

  __device__ void leave() {
    agent_ = nullptr;
  }

  __device__ void enter(Agent* agent);

  __device__ void decide();
};


class Agent {
 protected:
  Cell* position_;
  Cell* new_position_;
  uint32_t random_state_;
  uint8_t type_identifier_;

 public:
  __device__ Agent(uint32_t random_state, uint8_t type_identifier)
      : random_state_(random_state), type_identifier_(type_identifier) {}

  __device__ uint32_t& random_state() {
    return random_state_;
  }

  __device__ void set_new_position(Cell* new_pos) {
    new_position_ = new_pos;
  }

  __device__ Cell* position() const {
    return position_;
  }

  __device__ void set_position(Cell* cell) {
    position_ = cell;
  }

  // TODO: Verify that RTTI (dynamic_cast) does not work in device code.
  __device__ uint8_t type_identifier() const {
    return type_identifier_;
  }
};

__device__ void Cell::enter(Agent* agent) {
  agent_ = agent;
  agent->set_position(this);
}


class Fish : public Agent {
 private:
  uint32_t egg_timer_;

 public:
  static const uint8_t kTypeId = 1;

  __device__ Fish(uint32_t random_state) : Agent(random_state, kTypeId) {}

  __device__ void prepare() {
    egg_timer_++;
    // Fallback: Stay on current cell.
    new_position_ = position_;
    position_->request_random_free_neighbor(random_state_);
  }

  __device__ void update() {
    Cell* old_position = position_;

    if (old_position != new_position_) {
      old_position->leave();
      new_position_->enter(this);

      if (egg_timer_ > SPAWN_THRESHOLD) {
        old_position->enter(new Fish(random_state_ + 1));
        egg_timer_ = 0;
      }
    }
  }
};


class Shark : public Agent {
 private:
  uint32_t energy_;
  uint32_t egg_timer_;
  uint32_t random_state_;

 public:
  static const uint8_t kTypeId = 2;

  __device__ Shark(uint32_t random_state) : Agent(random_state, kTypeId) {}

  __device__ void prepare() {
    egg_timer_++;
    energy_--;

    if (energy_ == 0) {
      position_->kill();
    } else {
      // Fallback: Stay on current cell.
      new_position_ = position_;
      position_->request_random_fish_neighbor(random_state_);
    }
  }

  __device__ void update() {
    Cell* old_position = position_;

    if (old_position != new_position_) {
      if (new_position_->has_fish()) {
        energy_ += ENERGY_BOOST;
        new_position_->kill();
      }

      old_position->leave();
      new_position_->enter(this);

      if (egg_timer_ > SPAWN_THRESHOLD) {
        old_position->enter(new Shark(random_state_ + 1));
        egg_timer_ = 0;
      }
    }
  }
};

__device__ bool Cell::has_fish() const {
  return agent_ != nullptr && agent_->type_identifier() == Fish::kTypeId;
}

__device__ bool Cell::has_shark() const {
  return agent_ != nullptr && agent_->type_identifier() == Shark::kTypeId;
}

__device__ void Cell::kill() {
  delete agent_;
  leave();
}

__device__ void Cell::decide() {
  if (neighbor_request_[4]) {
    // This cell has priority.
  } else {
    uint8_t candidates[4];
    uint8_t num_candidates = 0;

    for (int i = 0; i < 4; ++i) {
      if (neighbor_request_[i]) {
        candidates[num_candidates++] = i;
      }
    }

    if (num_candidates != 0) {
      uint32_t selected_index = random_number(random_state_, num_candidates);
      uint8_t selected = candidates[selected_index];
      neighbors_[selected]->agent()->set_new_position(this);
    }
  }
}



__device__ Cell*  cells[GRID_SIZE_X * GRID_SIZE_Y];

__global__ void create_cells() {
  int tid = threadIdx.x + blockDim.x*blockIdx.x;

  if (tid < GRID_SIZE_Y*GRID_SIZE_X) {
    cells[tid] = new Cell(tid + 1);
  }
}

__global__ void setup_cells() {
  int tid = threadIdx.x + blockDim.x*blockIdx.x;

  if (tid < GRID_SIZE_Y*GRID_SIZE_X) {
    int x = tid % GRID_SIZE_X;
    int y = tid / GRID_SIZE_X;

    Cell* left = x > 0 ? cells[y*GRID_SIZE_X + x - 1] : nullptr;
    Cell* right = x < GRID_SIZE_X - 1 ? cells[y*GRID_SIZE_X + x + 1] : nullptr;
    Cell* top = y > 0 ? cells[(y - 1)*GRID_SIZE_X + x] : nullptr;
    Cell* bottom = y < GRID_SIZE_Y - 1 ? cells[(y + 1)*GRID_SIZE_X + x]
                                       : nullptr;

    // left, top, right, bottom
    cells[tid]->set_neighbors(left, top, right, bottom);

    // Initialize with random agent.

    uint32_t agent_type = random_number(cells[tid]->random_state(), 3);
    if (agent_type == 0) {
      cells[tid]->set_agent(new Fish(tid + 10001));
    } else if (agent_type == 1) {
      cells[tid]->set_agent(new Shark(tid + 20001));
    } else {
      // Free cell.
    }
  }
}

void initialize() {
  create_cells<<<GRID_SIZE_X*GRID_SIZE_Y/1024 + 1, 1024>>>();
  hipDeviceSynchronize();
  setup_cells<<<GRID_SIZE_X*GRID_SIZE_Y/1024 + 1, 1024>>>();
  hipDeviceSynchronize();
}

// Problem: It is not easy to keep track of all objects of a class if they are
// dynamically allocated. But we want to benchmark the performance of new/
// delete in CUDA.
// Solution: Fill these arrays in a separate kernel by iterating over all
// cells, storing agents in the respective array slots, and compacting the
// arrays. We do not measure the performance of these steps.
__device__ uint32_t num_sharks = 0;
__device__ Shark* sharks[GRID_SIZE_Y * GRID_SIZE_X];
__device__ uint32_t num_fish = 0;
__device__ Fish*  fish[GRID_SIZE_Y * GRID_SIZE_X];

__global__ void reset_agent_arrays() {
  num_sharks = 0;
  num_fish = 0;
}

// One thread per cell.
// TODO: Reset counters to zero before running the kernel.
__global__ void find_agents() {
  int tid = threadIdx.x + blockDim.x*blockIdx.x;

  if (tid < GRID_SIZE_Y*GRID_SIZE_X) {
    if (cells[tid]->has_shark()) {
      uint32_t idx = atomicAdd(&num_sharks, 1);
      sharks[idx] = reinterpret_cast<Shark*>(cells[tid]->agent());
    } else if (cells[tid]->has_fish()) {
      uint32_t idx = atomicAdd(&num_fish, 1);
      fish[idx] = reinterpret_cast<Fish*>(cells[tid]->agent());
    }
  }
}

void generate_agent_arrays() {
  reset_agent_arrays<<<1, 1>>>();
  hipDeviceSynchronize();
  find_agents<<<GRID_SIZE_X*GRID_SIZE_Y/1024 + 1, 1024>>>();
  hipDeviceSynchronize();
}

__device__ uint32_t d_gui_map[GRID_SIZE_Y * GRID_SIZE_X];
uint32_t gui_map[GRID_SIZE_Y * GRID_SIZE_X];

__global__ void fill_gui_map() {
  int tid = threadIdx.x + blockDim.x*blockIdx.x;

  if (tid < GRID_SIZE_Y*GRID_SIZE_X) {
    if (cells[tid]->agent() != nullptr) {
      d_gui_map[tid] = cells[tid]->agent()->type_identifier();
    }
  }
}

void update_gui_map() {
  fill_gui_map<<<GRID_SIZE_X*GRID_SIZE_Y/1024 + 1, 1024>>>();
  hipDeviceSynchronize();

  hipMemcpy(gui_map, d_gui_map, sizeof(uint32_t)*GRID_SIZE_X*GRID_SIZE_Y,
             hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
}

// SDL helper variables.
SDL_Window* window_;
SDL_Renderer* renderer_;


int main(int argc, char* arvg[]) {
  // Initialize renderer.
  if (SDL_Init(SDL_INIT_VIDEO)) {
    printf("SDL_Init Error: %s", SDL_GetError());
    exit(1);
  }

  window_ = SDL_CreateWindow("Wa-Tor", 100, 100,
                             GRID_SIZE_X, GRID_SIZE_Y, SDL_WINDOW_OPENGL);
  if (window_ == NULL) { 
    printf("SDL_CreateWindow Error: %s", SDL_GetError());
    SDL_Quit();
    exit(2);
  }

  renderer_ = SDL_CreateRenderer(window_, -1,
      SDL_RENDERER_ACCELERATED | SDL_RENDERER_PRESENTVSYNC);
  if (renderer_ == NULL) { 
    SDL_DestroyWindow(window_);
    printf("SDL_CreateRenderer Error: %s", SDL_GetError());
    SDL_Quit();
    exit(3);
  }

  // Draw black background.
  SDL_SetRenderDrawColor(renderer_, 0, 0, 0, 0);
  SDL_RenderClear(renderer_);
  SDL_RenderPresent(renderer_);
}