#include "hip/hip_runtime.h"
#include <hiprand/hiprand_kernel.h>
#include <limits>
#include <algorithm>
#include <random>

#include "allocator/soa_allocator.h"
#include "allocator/soa_base.h"
#include "allocator/allocator_handle.h"

#include "configuration.h"

static const int kIntMax = std::numeric_limits<int>::max();

// Pre-declare all classes.
class C1;
class C2;

// Declare allocator type.
using AllocatorT = SoaAllocator<16*64*64*64*64, C1, C2>;

// Allocator handles.
__device__ AllocatorT* device_allocator;
AllocatorHandle<AllocatorT>* allocator_handle;

__device__ unsigned long long int d_checksum;

// 32 byte objects.
class C1 : public SoaBase<AllocatorT> {
 public:
  declare_field_types(C1, C2*, int, int, int, int, int, int)

  SoaField<C1, 0> other_;
  SoaField<C1, 1> id_;
  SoaField<C1, 2> rand_num_;
  SoaField<C1, 3> int3_;
  SoaField<C1, 4> int4_;
  SoaField<C1, 5> int5_;
  SoaField<C1, 6> int6_;

  __device__ C1(int id, int rand_num)
      : id_(id), rand_num_(rand_num), other_(nullptr) {}

  __device__ void compute_checksum();
};

// 32 byte objects.
class C2 : public SoaBase<AllocatorT> {
 public:
  declare_field_types(C2, C1*, int, int, int, int, int, int)

  SoaField<C2, 0> other_;
  SoaField<C2, 1> id_;
  SoaField<C2, 2> rand_num_;
  SoaField<C2, 3> int3_;
  SoaField<C2, 4> int4_;
  SoaField<C2, 5> int5_;
  SoaField<C2, 6> int6_;

  __device__ C2(int id, int rand_num)
      : id_(id), rand_num_(rand_num), other_(nullptr) {}

  __device__ void maybe_destroy_object() {
    if (rand_num_ % kDeleteFactor == 0) {
      if (other_ != nullptr) {
        other_->other_ = nullptr;
      }

      destroy(device_allocator, this);
    }
  }
};

__device__ void C1::compute_checksum() {
  if (other_ != nullptr) {
    atomicAdd(&d_checksum,  (id_ * other_->id_) % 97);
  }
}

__global__ void kernel_create_objects(C1** ptr_c1, C2** ptr_c2) {
  hiprandState_t random_state;
  hiprand_init(43, threadIdx.x + blockDim.x*blockIdx.x,
              0, &random_state);

  for (int i = threadIdx.x + blockDim.x*blockIdx.x;
       i < kSize; i += blockDim.x * gridDim.x) {
    ptr_c1[i] = new(device_allocator) C1(i, hiprand(&random_state) % kIntMax);
    ptr_c2[i] = new(device_allocator) C2(i, hiprand(&random_state) % kIntMax);
  }
}

size_t h_assoc[kSize];

__global__ void kernel_set_pointers(C1** ptr_c1, C2** ptr_c2,
                                    size_t* d_assoc) {
  hiprandState_t random_state;
  hiprand_init(42, threadIdx.x + blockDim.x*blockIdx.x,
              0, &random_state);

  for (int i = threadIdx.x + blockDim.x*blockIdx.x;
       i < kSize; i += blockDim.x * gridDim.x) {
    ptr_c2[i]->other_ = ptr_c1[d_assoc[i]];
    ptr_c1[d_assoc[i]]->other_ = ptr_c2[i];
  }
}

void set_pointers(C1** ptr_c1, C2** ptr_c2) {
  for (size_t i = 0; i < kSize; ++i) {
    h_assoc[i] = i;
  }
  shuffle(h_assoc, h_assoc + kSize, std::default_random_engine(42));

  size_t* d_assoc;
  hipMalloc(&d_assoc, sizeof(size_t)*kSize);
  hipMemcpy(d_assoc, h_assoc, sizeof(size_t)*kSize,
             hipMemcpyHostToDevice);

  kernel_set_pointers<<<512, 512>>>(ptr_c1, ptr_c2, d_assoc);
  gpuErrchk(hipDeviceSynchronize());
}

int main(int /*argc*/, char** /*argv*/) {
  // Create new allocator.
  allocator_handle = new AllocatorHandle<AllocatorT>();
  AllocatorT* dev_ptr = allocator_handle->device_pointer();
  hipMemcpyToSymbol(HIP_SYMBOL(device_allocator), &dev_ptr, sizeof(AllocatorT*), 0,
                     hipMemcpyHostToDevice);

  C1** d_ptr_c1;
  C2** d_ptr_c2;
  gpuErrchk(hipMalloc(&d_ptr_c1, sizeof(C1*)*kSize));
  gpuErrchk(hipMalloc(&d_ptr_c2, sizeof(C2*)*kSize));

  // Create objects.
  kernel_create_objects<<<512, 512>>>(d_ptr_c1, d_ptr_c2);
  gpuErrchk(hipDeviceSynchronize());

  set_pointers(d_ptr_c1, d_ptr_c2);

  // Destroy some objects.
  allocator_handle->parallel_do<C2, &C2::maybe_destroy_object>();
  gpuErrchk(hipDeviceSynchronize());

  int total_time = 0;
  auto time_before = std::chrono::system_clock::now();

#ifdef OPTION_DEFRAG
  // Defragment C2.
  allocator_handle->parallel_defrag<C2>();
#endif  // OPTION_DEFRAG

  auto time_after = std::chrono::system_clock::now();
  int time_running = std::chrono::duration_cast<std::chrono::milliseconds>(
      time_after - time_before).count();
  total_time += time_running;

#ifdef OPTION_DEFRAG
  allocator_handle->DBG_print_defrag_time();
  allocator_handle->DBG_collect_stats();
  allocator_handle->DBG_print_collected_stats();
#endif  // OPTION_DEFRAG

  // Compute checksum.
  unsigned long long int h_checksum = 0;
  hipMemcpyToSymbol(HIP_SYMBOL(d_checksum), &h_checksum, sizeof(unsigned long long int),
                     0, hipMemcpyHostToDevice);

  allocator_handle->parallel_do<C1, &C1::compute_checksum>();
  gpuErrchk(hipDeviceSynchronize());

  hipMemcpyFromSymbol(&h_checksum, HIP_SYMBOL(d_checksum), sizeof(unsigned long long int),
                       0, hipMemcpyDeviceToHost);
  printf("Checksum: %llu\n", h_checksum);

  printf("%i\n", total_time);
}
