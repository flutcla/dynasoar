#include "hip/hip_runtime.h"
#include <assert.h>
#include <chrono>
#include <hiprand/hiprand_kernel.h>
#include <limits>
#include <stdio.h>

#include "configuration.h"
#include "dataset.h"
#include "util/util.h"
#include "rendering.h"


static const int kThreads = 256;
static const int kNullptr = std::numeric_limits<int>::max();

using IndexT = int;

__device__ DeviceArray<IndexT, kMaxDegree>* dev_Node_springs;
__device__ float* dev_Node_pos_x;
__device__ float* dev_Node_pos_y;
__device__ int* dev_Node_num_springs;
__device__ float* dev_Node_vel_x;
__device__ float* dev_Node_vel_y;
__device__ float* dev_Node_mass;
__device__ char* dev_Node_type;
__device__ IndexT* dev_Spring_p1;
__device__ IndexT* dev_Spring_p2;
__device__ float* dev_Spring_factor;
__device__ float* dev_Spring_initial_length;
__device__ float* dev_Spring_force;
__device__ float* dev_Spring_max_force;
__device__ bool* dev_Spring_is_active;


__device__ void new_NodeBase(IndexT id, float pos_x, float pos_y) {
  dev_Node_pos_x[id] = pos_x;
  dev_Node_pos_y[id] = pos_y;
  dev_Node_num_springs[id] = 0;
  dev_Node_type[id] = kTypeNodeBase;
}


__device__ void new_AnchorNode(IndexT id, float pos_x, float pos_y) {
  new_NodeBase(id, pos_x, pos_y);
  dev_Node_type[id] = kTypeAnchorNode;
}


__device__ void new_AnchorPullNode(IndexT id, float pos_x, float pos_y,
                                   float vel_x, float vel_y) {
  new_AnchorNode(id, pos_x, pos_y);
  dev_Node_vel_x[id] = vel_x;
  dev_Node_vel_y[id] = vel_y;
  dev_Node_type[id] = kTypeAnchorPullNode;
}


__device__ void new_Node(IndexT id, float pos_x, float pos_y, float mass) {
  new_NodeBase(id, pos_x, pos_y);
  dev_Node_mass[id] = mass;
  dev_Node_type[id] = kTypeNode;
}


__device__ float NodeBase_distance_to(IndexT id, IndexT other) {
  float dx = dev_Node_pos_x[id] - dev_Node_pos_x[other];
  float dy = dev_Node_pos_y[id] - dev_Node_pos_y[other];
  float dist_sq = dx*dx + dy*dy;
  return sqrt(dist_sq);
}


__device__ void NodeBase_add_spring(IndexT id, IndexT spring) {
  assert(id >= 0 && id < kMaxNodes);

  int idx = atomicAdd(&dev_Node_num_springs[id], 1);
  assert(idx + 1 <= kMaxDegree);
  dev_Node_springs[id][idx] = spring;

  assert(dev_Spring_p1[spring] == id || dev_Spring_p2[spring] == id);
}


__device__ void new_Spring(IndexT id, IndexT p1, IndexT p2,
                           float spring_factor, float max_force) {
  dev_Spring_is_active[id] = true;
  dev_Spring_p1[id] = p1;
  dev_Spring_p2[id] = p2;
  dev_Spring_factor[id] = spring_factor;
  dev_Spring_force[id] = 0.0f;
  dev_Spring_max_force[id] = max_force;
  dev_Spring_initial_length[id] = NodeBase_distance_to(p1, p2);
  assert(dev_Spring_initial_length[id] > 0.0f);

  NodeBase_add_spring(p1, id);
  NodeBase_add_spring(p2, id);
}


__device__ void NodeBase_remove_spring(IndexT id, IndexT spring) {
  // TODO: This won't work if two springs break at the same time.

  int i = 0;
  IndexT s = kNullptr;

  do {
    assert(i < kMaxDegree);
    s = dev_Node_springs[id][i];
    ++i;
  } while(s != spring);

  for (; i < dev_Node_num_springs[id]; ++i) {
    dev_Node_springs[id][i - 1] = dev_Node_springs[id][i];
  }

  --dev_Node_num_springs[id];

  if (dev_Node_num_springs[id] == 0) {
    dev_Node_type[id] = 0;
  }
}


__device__ void AnchorPullNode_pull(IndexT id) {
  dev_Node_pos_x[id] += dev_Node_vel_x[id] * kDt;
  dev_Node_pos_y[id] += dev_Node_vel_y[id] * kDt;
}


__device__ void Spring_compute_force(IndexT id) {
  float dist = NodeBase_distance_to(dev_Spring_p1[id], dev_Spring_p2[id]);
  float displacement = max(0.0f, dist - dev_Spring_initial_length[id]);
  dev_Spring_force[id] = dev_Spring_factor[id] * displacement;

  if (dev_Spring_force[id] > dev_Spring_max_force[id]) {
    NodeBase_remove_spring(dev_Spring_p1[id], id);
    NodeBase_remove_spring(dev_Spring_p2[id], id);
    dev_Spring_is_active[id] = false;
  }
}


__device__ void Node_move(IndexT id) {
  float force_x = 0.0f;
  float force_y = 0.0f;

  for (int i = 0; i < dev_Node_num_springs[id]; ++i) {
    IndexT s = dev_Node_springs[id][i];
    IndexT from;
    IndexT to;

    if (dev_Spring_p1[s] == id) {
      from = id;
      to = dev_Spring_p2[s];
    } else {
      assert(dev_Spring_p2[s] == id);
      from = id;
      to = dev_Spring_p1[s];
    }

    // Calculate unit vector.
    float dx = dev_Node_pos_x[to] - dev_Node_pos_x[from];
    float dy = dev_Node_pos_y[to] - dev_Node_pos_y[from];
    float dist = sqrt(dx*dx + dy*dy);
    float unit_x = dx/dist;
    float unit_y = dy/dist;

    // Apply force.
    force_x += unit_x*dev_Spring_force[s];
    force_y += unit_y*dev_Spring_force[s];
  }

  // Calculate new velocity and position.
  dev_Node_vel_x[id] += force_x*kDt / dev_Node_mass[id];
  dev_Node_vel_y[id] += force_y*kDt / dev_Node_mass[id];
  dev_Node_vel_x[id] *= 1.0f - kVelocityDampening;
  dev_Node_vel_y[id] *= 1.0f - kVelocityDampening;
  dev_Node_pos_x[id] += dev_Node_vel_x[id]*kDt;
  dev_Node_pos_y[id] += dev_Node_vel_y[id]*kDt;
}


// Only for rendering.
__device__ int dev_num_springs;
__device__ SpringInfo dev_spring_info[kMaxSprings];
int host_num_springs;
SpringInfo host_spring_info[kMaxSprings];

__device__ void Spring_add_to_rendering_array(IndexT id) {
  int idx = atomicAdd(&dev_num_springs, 1);
  dev_spring_info[idx].p1_x = dev_Node_pos_x[dev_Spring_p1[id]];
  dev_spring_info[idx].p1_y = dev_Node_pos_y[dev_Spring_p1[id]];
  dev_spring_info[idx].p2_x = dev_Node_pos_x[dev_Spring_p2[id]];
  dev_spring_info[idx].p2_y = dev_Node_pos_y[dev_Spring_p2[id]];
  dev_spring_info[idx].force = dev_Spring_force[id];
  dev_spring_info[idx].max_force = dev_Spring_max_force[id];
}


__global__ void kernel_AnchorPullNode_pull() {
  for (int i = threadIdx.x + blockDim.x * blockIdx.x;
       i < kMaxNodes; i += blockDim.x * gridDim.x) {
    if (dev_Node_type[i] == kTypeAnchorPullNode) {
      AnchorPullNode_pull(i);
    }
  }
}


__global__ void kernel_Node_move() {
  for (int i = threadIdx.x + blockDim.x * blockIdx.x;
       i < kMaxNodes; i += blockDim.x * gridDim.x) {
    if (dev_Node_type[i] == kTypeNode) {
      Node_move(i);
    }
  }
}


__global__ void kernel_Spring_compute_force() {
  for (int i = threadIdx.x + blockDim.x * blockIdx.x;
       i < kMaxSprings; i += blockDim.x * gridDim.x) {
    if (dev_Spring_is_active[i]) {
      Spring_compute_force(i);
    }
  }
}


__global__ void kernel_Spring_add_to_rendering_array() {
  for (int i = threadIdx.x + blockDim.x * blockIdx.x;
       i < kMaxSprings; i += blockDim.x * gridDim.x) {
    if (dev_Spring_is_active[i]) {
      Spring_add_to_rendering_array(i);
    }
  }
}


__global__ void kernel_initialize_nodes() {
  for (int i = threadIdx.x + blockDim.x * blockIdx.x;
       i < kMaxNodes; i += blockDim.x * gridDim.x) {
    dev_Node_type[i] = 0;
  }
}


__global__ void kernel_initialize_springs() {
  for (int i = threadIdx.x + blockDim.x * blockIdx.x;
       i < kMaxSprings; i += blockDim.x * gridDim.x) {
    dev_Spring_is_active[i] = false;
  }
}


void transfer_data() {
  int zero = 0;
  hipMemcpyToSymbol(HIP_SYMBOL(dev_num_springs), &zero, sizeof(int), 0,
                     hipMemcpyHostToDevice);
  gpuErrchk(hipDeviceSynchronize());

  kernel_Spring_add_to_rendering_array<<<128, 128>>>();
  gpuErrchk(hipDeviceSynchronize());

  hipMemcpyFromSymbol(&host_num_springs, HIP_SYMBOL(dev_num_springs), sizeof(int), 0,
                       hipMemcpyDeviceToHost);
  gpuErrchk(hipDeviceSynchronize());

  hipMemcpyFromSymbol(host_spring_info, HIP_SYMBOL(dev_spring_info),
                       sizeof(SpringInfo)*host_num_springs, 0,
                       hipMemcpyDeviceToHost);
  gpuErrchk(hipDeviceSynchronize());
}


float checksum() {
  transfer_data();
  float result = 0.0f;

  for (int i = 0; i < host_num_springs; ++i) {
    result += host_spring_info[i].p1_x*host_spring_info[i].p2_y
              *host_spring_info[i].force;
  }

  return result;
}


void compute() {
  kernel_Spring_compute_force<<<(kMaxSprings + kThreads - 1) / kThreads,
                                kThreads>>>();
  gpuErrchk(hipDeviceSynchronize());

  kernel_Node_move<<<(kMaxNodes + kThreads - 1) / kThreads,
                     kThreads>>>();
  gpuErrchk(hipDeviceSynchronize());
}


void step() {
  kernel_AnchorPullNode_pull<<<(kMaxNodes + kThreads - 1) / kThreads,
                               kThreads>>>();
  gpuErrchk(hipDeviceSynchronize());

  for (int i = 0; i < kNumComputeIterations; ++i) {
    compute();
  }
}


void initialize_memory() {
  kernel_initialize_nodes<<<128, 128>>>();
  gpuErrchk(hipDeviceSynchronize());

  kernel_initialize_springs<<<128, 128>>>();
  gpuErrchk(hipDeviceSynchronize());
}


//__device__ NodeBase* tmp_nodes[kMaxNodes];

__global__ void kernel_create_nodes(DsNode* nodes, int num_nodes) {
  for (int i = threadIdx.x + blockDim.x * blockIdx.x;
       i < num_nodes; i += blockDim.x * gridDim.x) {
    if (nodes[i].type == kTypeNode) {
      new_Node(i, nodes[i].pos_x, nodes[i].pos_y, nodes[i].mass);
    } else if (nodes[i].type == kTypeAnchorPullNode) {
      new_AnchorPullNode(i, nodes[i].pos_x, nodes[i].pos_y, nodes[i].vel_x,
                         nodes[i].vel_y);
    } else {
      assert(false);
    }
  }
}


__global__ void kernel_create_springs(DsSpring* springs, int num_springs) {
  for (int i = threadIdx.x + blockDim.x * blockIdx.x;
       i < num_springs; i += blockDim.x * gridDim.x) {
    new_Spring(i, springs[i].p1, springs[i].p2,
               springs[i].spring_factor, springs[i].max_force);
  }
}


void load_dataset(Dataset& dataset) {
  DsNode* host_nodes;
  hipMalloc(&host_nodes, sizeof(DsNode)*dataset.nodes.size());
  hipMemcpy(host_nodes, dataset.nodes.data(),
             sizeof(DsNode)*dataset.nodes.size(), hipMemcpyHostToDevice);

  DsSpring* host_springs;
  hipMalloc(&host_springs, sizeof(DsSpring)*dataset.springs.size());
  hipMemcpy(host_springs, dataset.springs.data(),
             sizeof(DsSpring)*dataset.springs.size(), hipMemcpyHostToDevice);
  gpuErrchk(hipDeviceSynchronize());

  kernel_create_nodes<<<128, 128>>>(host_nodes, dataset.nodes.size());
  gpuErrchk(hipDeviceSynchronize());

  kernel_create_springs<<<128, 128>>>(host_springs, dataset.springs.size());
  gpuErrchk(hipDeviceSynchronize());

  hipFree(host_nodes);
  hipFree(host_springs);
}


int main(int /*argc*/, char** /*argv*/) {
  // Allocate memory.
  DeviceArray<IndexT, kMaxDegree>* host_Node_springs;
  hipMalloc(&host_Node_springs,
             sizeof(DeviceArray<IndexT, kMaxDegree>)*kMaxNodes);
  hipMemcpyToSymbol(HIP_SYMBOL(dev_Node_springs), &host_Node_springs,
                     sizeof(DeviceArray<IndexT, kMaxDegree>*), 0,
                     hipMemcpyHostToDevice);

  float* host_Node_pos_x;
  hipMalloc(&host_Node_pos_x, sizeof(float)*kMaxNodes);
  hipMemcpyToSymbol(HIP_SYMBOL(dev_Node_pos_x), &host_Node_pos_x, sizeof(float*), 0,
                     hipMemcpyHostToDevice);

  float* host_Node_pos_y;
  hipMalloc(&host_Node_pos_y, sizeof(float)*kMaxNodes);
  hipMemcpyToSymbol(HIP_SYMBOL(dev_Node_pos_y), &host_Node_pos_y, sizeof(float*), 0,
                     hipMemcpyHostToDevice);

  float* host_Node_vel_x;
  hipMalloc(&host_Node_vel_x, sizeof(float)*kMaxNodes);
  hipMemcpyToSymbol(HIP_SYMBOL(dev_Node_vel_x), &host_Node_vel_x, sizeof(float*), 0,
                     hipMemcpyHostToDevice);

  float* host_Node_vel_y;
  hipMalloc(&host_Node_vel_y, sizeof(float)*kMaxNodes);
  hipMemcpyToSymbol(HIP_SYMBOL(dev_Node_vel_y), &host_Node_vel_y, sizeof(float*), 0,
                     hipMemcpyHostToDevice);

  int* host_Node_num_springs;
  hipMalloc(&host_Node_num_springs, sizeof(int)*kMaxNodes);
  hipMemcpyToSymbol(HIP_SYMBOL(dev_Node_num_springs), &host_Node_num_springs,
                     sizeof(int*), 0, hipMemcpyHostToDevice);

  float* host_Node_mass;
  hipMalloc(&host_Node_mass, sizeof(float)*kMaxNodes);
  hipMemcpyToSymbol(HIP_SYMBOL(dev_Node_mass), &host_Node_mass, sizeof(float*), 0,
                     hipMemcpyHostToDevice);

  char* host_Node_type;
  hipMalloc(&host_Node_type, sizeof(char)*kMaxNodes);
  hipMemcpyToSymbol(HIP_SYMBOL(dev_Node_type), &host_Node_type, sizeof(char*), 0,
                     hipMemcpyHostToDevice);

  IndexT* host_Spring_p1;
  hipMalloc(&host_Spring_p1, sizeof(IndexT)*kMaxSprings);
  hipMemcpyToSymbol(HIP_SYMBOL(dev_Spring_p1), &host_Spring_p1, sizeof(IndexT*), 0,
                     hipMemcpyHostToDevice);

  IndexT* host_Spring_p2;
  hipMalloc(&host_Spring_p2, sizeof(IndexT)*kMaxSprings);
  hipMemcpyToSymbol(HIP_SYMBOL(dev_Spring_p2), &host_Spring_p2, sizeof(IndexT*), 0,
                     hipMemcpyHostToDevice);

  float* host_Spring_factor;
  hipMalloc(&host_Spring_factor, sizeof(float)*kMaxSprings);
  hipMemcpyToSymbol(HIP_SYMBOL(dev_Spring_factor), &host_Spring_factor, sizeof(float*), 0,
                     hipMemcpyHostToDevice);

  float* host_Spring_initial_length;
  hipMalloc(&host_Spring_initial_length, sizeof(float)*kMaxSprings);
  hipMemcpyToSymbol(HIP_SYMBOL(dev_Spring_initial_length), &host_Spring_initial_length,
                     sizeof(float*), 0, hipMemcpyHostToDevice);

  float* host_Spring_force;
  hipMalloc(&host_Spring_force, sizeof(float)*kMaxSprings);
  hipMemcpyToSymbol(HIP_SYMBOL(dev_Spring_force), &host_Spring_force, sizeof(float*), 0,
                     hipMemcpyHostToDevice);

  float* host_Spring_max_force;
  hipMalloc(&host_Spring_max_force, sizeof(float)*kMaxSprings);
  hipMemcpyToSymbol(HIP_SYMBOL(dev_Spring_max_force), &host_Spring_max_force,
                     sizeof(float*), 0, hipMemcpyHostToDevice);

  bool* host_Spring_is_active;
  hipMalloc(&host_Spring_is_active, sizeof(bool)*kMaxSprings);
  hipMemcpyToSymbol(HIP_SYMBOL(dev_Spring_is_active), &host_Spring_is_active,
                     sizeof(bool*), 0, hipMemcpyHostToDevice);

  initialize_memory();

  //load_example<<<1, 1>>>();
  //load_random<<<1, 1>>>();

  Dataset dataset;
  random_dataset(dataset);
  load_dataset(dataset);

  auto time_start = std::chrono::system_clock::now();

  for (int i = 0; i < kNumSteps; ++i) {
    step();
  }

  auto time_end = std::chrono::system_clock::now();
  auto elapsed = time_end - time_start;
  auto millis = std::chrono::duration_cast<std::chrono::milliseconds>(elapsed)
      .count();

  printf("Time: %lu ms\n", millis);
  printf("Checksum: %f\n", checksum());
}
