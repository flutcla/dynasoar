#include "hip/hip_runtime.h"

#include "configuration.h"
#include "rendering.h"
#include "traffic.h"


static const int kNumBlockSize = 256;

// TODO: Consider migrating to SoaAlloc.
TrafficLight* h_traffic_lights;
__device__ TrafficLight* d_traffic_lights;
Node* h_nodes;
__device__ Node* d_nodes;


// Only for rendering.
__device__ int dev_num_cells;
__device__ float* dev_Cell_pos_x;
__device__ float* dev_Cell_pos_y;
__device__ bool* dev_Cell_occupied;
float* host_Cell_pos_x;
float* host_Cell_pos_y;
bool* host_Cell_occupied;
float* host_data_Cell_pos_x;
float* host_data_Cell_pos_y;
bool* host_data_Cell_occupied;
int host_num_cells;


// Allocator handles.
__device__ AllocatorT* device_allocator;
AllocatorHandle<AllocatorT>* allocator_handle;


__device__ void Cell::occupy(Car* car) {
  assert(is_free());
  car_ = car;
}


__device__ void Cell::release() {
  assert(!is_free());
  car_ = nullptr;
}


__device__ void Car::step_prepare_path() {
  step_initialize_iteration();
  step_accelerate();
  step_extend_path();
  step_constraint_velocity();
  step_slow_down();
}


__device__ Cell* Car::next_step(Cell* position) {
  // Almost random walk.
  const uint32_t num_outgoing = position->num_outgoing();
  assert(num_outgoing > 0);

  // Need some kind of return statement here.
  return position->get_outgoing(random_int(0, num_outgoing));
}


__device__ void Car::step_initialize_iteration() {
  // Reset calculated path. This forces cars with a random moving behavior to
  // select a new path in every iteration. Otherwise, cars might get "stuck"
  // on a full network if many cars are waiting for the one in front of them in
  // a cycle.
  path_length_ = 0;
}


__device__ void Car::step_accelerate() {
  // Speed up the car by 1 or 2 units.
  int speedup = random_int(0, 2) + 1;
  velocity_ = max_velocity_ < velocity_ + speedup
      ? max_velocity_ : velocity_ + speedup;
}


__device__ void Car::step_extend_path() {
  Cell* cell = position_;
  Cell* next_cell;

  for (int i = 0; i < velocity_; ++i) {
    if (cell->is_sink()) {
      break;
    }

    next_cell = next_step(cell);
    assert(next_cell != cell);

    cell = next_cell;
    path_[i] = cell;
    path_length_ = path_length_ + 1;
  }

  velocity_ = path_length_;
}


__device__ void Car::step_constraint_velocity() {
  // This is actually only needed for the very first iteration, because a car
  // may be positioned on a traffic light cell.
  if (velocity_ > position()->max_velocity()) {
    velocity_ = position()->max_velocity();
  }

  int path_index = 0;
  int distance = 1;

  while (distance <= velocity_) {
    // Invariant: Movement of up to `distance - 1` many cells at `velocity_`
    //            is allowed.
    // Now check if next cell can be entered.
    Cell* next_cell = path_[path_index];

    // Avoid collision.
    if (!next_cell->is_free()) {
      // Cannot enter cell.
      --distance;
      velocity_ = distance;
      break;
    } // else: Can enter next cell.

    if (velocity_ > next_cell->max_velocity()) {
      // Car is too fast for this cell.
      if (next_cell->max_velocity() > distance - 1) {
        // Even if we slow down, we would still make progress.
        velocity_ = next_cell->max_velocity();
      } else {
        // Do not enter the next cell.
        --distance;
        assert(distance >= 0);

        velocity_ = distance;
        break;
      }
    }

    ++distance;
    ++path_index;
  }

  --distance;

#ifndef NDEBUG
  for (int i = 0; i < velocity_; ++i) {
    assert(path_[i]->is_free());
    assert(i == 0 || path_[i - 1] != path_[i]);
  }
  // TODO: Check why the cast is necessary.
  assert(distance <= velocity());
#endif  // NDEBUG
}


__device__ void Car::step_move() {
  Cell* cell = position_;
  for (int i = 0; i < velocity_; ++i) {
    assert(path_[i] != cell);

    cell = path_[i];
    assert(cell->is_free());

    position()->release();
    cell->occupy(this);
    position_ = cell;
  }

  if (position()->is_sink() || position()->is_target()) {
    // Remove car from the simulation. Will be added again in the next
    // iteration.
    position()->release();
    device_allocator->free(this);
  }
}


__device__ void Car::step_slow_down() {
  // 20% change of slowdown.
  if (hiprand_uniform(&random_state_) < 0.2 && velocity_ > 0) {
    velocity_ = velocity_ - 1;
  }
}


__device__ void TrafficLight::step() {
  if (num_cells_ > 0) {
    timer_ = (timer_ + 1) % phase_time_;

    if (timer_ == 0) {
      assert(cells_[phase_] != nullptr);
      cells_[phase_]->set_current_max_velocity(0);
      phase_ = (phase_ + 1) % num_cells_;
      cells_[phase_]->remove_speed_limit();
    }
  }
}


__device__ void ProducerCell::create_car() {
  if (is_free()) {
    float r = hiprand_uniform(&random_state_);
    if (r < kCarAllocationRatio) {
      Car* new_car = device_allocator->make_new<Car>(
          /*seed=*/ hiprand(&random_state_), /*cell=*/ this,
          /*max_velocity=*/ hiprand(&random_state_) % (kMaxVelocity/2)
                            + kMaxVelocity/2);
    }
  }
}


__device__ Car::Car(int seed, Cell* cell, int max_velocity)
    : position_(cell), path_length_(0), velocity_(0),
      max_velocity_(max_velocity) {
  cell->occupy(this);
  hiprand_init(seed, 0, 0, &random_state_);
}


__device__ Cell::Cell(int max_velocity, float x, float y)
    : car_(nullptr), max_velocity_(max_velocity),
      current_max_velocity_(max_velocity),
      num_incoming_(0), num_outgoing_(0), x_(x), y_(y), is_target_(false) {
  atomicAdd(&dev_num_cells, 1);
}


__device__ void Cell::add_to_rendering_array() {
  int idx = atomicAdd(&dev_num_cells, 1);
  dev_Cell_pos_x[idx] = x_;
  dev_Cell_pos_y[idx] = y_;
  dev_Cell_occupied[idx] = !is_free();
}


__global__ void kernel_traffic_light_step() {
  for (int i = blockIdx.x * blockDim.x + threadIdx.x;
       i < kNumIntersections; i += blockDim.x * gridDim.x) {
    d_traffic_lights[i].step();
  }
}


__global__ void kernel_create_nodes() {
  for (int i = blockIdx.x * blockDim.x + threadIdx.x;
       i < kNumIntersections; i += blockDim.x * gridDim.x) {
    hiprandState_t state;
    hiprand_init(i, 0, 0, &state);

    d_nodes[i].num_edges = hiprand(&state) % kMaxDegree + 1;
    d_nodes[i].num_incoming = 0;
    float x = hiprand_uniform(&state);
    float y = hiprand_uniform(&state);
    d_nodes[i].x = x;
    d_nodes[i].y = y;

    for (int j = 0; j < d_nodes[i].num_edges; ++j) {
      d_nodes[i].cell_out[j] = device_allocator->make_new<Cell>(
          /*max_velocity=*/ hiprand(&state) % (kMaxVelocity/2)
                            + kMaxVelocity/2,
          x, y);
    }
  }
}


__global__ void kernel_create_edges() {
  for (int i = blockIdx.x * blockDim.x + threadIdx.x;
       i < kNumIntersections; i += blockDim.x * gridDim.x) {
    hiprandState_t state;
    hiprand_init(i, 0, 0, &state);

    for (int k = 0; k < d_nodes[i].num_edges; ++k) {
      int target = -1;
      while (true) {
        target = hiprand(&state) % kNumIntersections;
        int num_in = d_nodes[i].num_incoming;

        if (num_in < kMaxDegree) {
          // Try...
          if (atomicCAS(&d_nodes[i].num_incoming, num_in, num_in + 1) == num_in) {
            printf("Connect: %i --> %i\n", i, target);
            // Create edge.
            float dx = d_nodes[i].x - d_nodes[target].x;
            float dy = d_nodes[i].y - d_nodes[target].y;
            float dist = sqrt(dx*dx + dy*dy);
            int steps = dist/kCellLength;
            float step_x = dx/steps;
            float step_y = dy/steps;
            Cell* prev = d_nodes[i].cell_out[k];

            for (int j = 0; j < steps; ++j) {
              float new_x = d_nodes[i].x + j*step_x;
              float new_y = d_nodes[i].y + j*step_y;
              Cell* next;

              if (hiprand_uniform(&state) < kProducerRatio) {
                next = device_allocator->make_new<ProducerCell>(
                    /*max_velocity=*/ prev->max_velocity(), new_x, new_y,
                    hiprand(&state));
              } else {
                next = device_allocator->make_new<Cell>(
                    /*max_velocity=*/ prev->max_velocity(), new_x, new_y);
              }

              if (hiprand_uniform(&state) < kTargetRatio) {
                next->set_target();
              }

              prev->set_num_outgoing(1);
              prev->set_outgoing(0, next);
              next->set_num_incoming(1);
              next->set_incoming(0, prev);

              prev = next;
            }

            // Connect to all outgoing nodes.
            prev->set_num_outgoing(d_nodes[target].num_edges);
            for (int j = 0; j < d_nodes[target].num_edges; ++j) {
              prev->set_outgoing(j, d_nodes[target].cell_out[j]);
              d_nodes[target].cell_out[j]->set_incoming(num_in, prev);
            }
            d_nodes[target].cell_in[num_in] = prev;
          }
        }
      }
    }
  }
}


__global__ void kernel_create_traffic_lights() {
  for (int i = blockIdx.x * blockDim.x + threadIdx.x;
       i < kNumIntersections; i += blockDim.x * gridDim.x) {
    new(d_traffic_lights + i) TrafficLight(
        /*num_cells=*/ d_nodes[i].num_incoming,
        /*phase_time=*/ 5);

    for (int j = 0; j < d_nodes[i].num_edges; ++j) {
      d_nodes[i].cell_out[j]->set_num_incoming(d_nodes[i].num_incoming);
    }

    for (int j = 0; j < d_nodes[i].num_incoming; ++j) {
      d_traffic_lights[i].set_cell(j, d_nodes[i].cell_in[j]);
      d_nodes[i].cell_in[j]->set_current_max_velocity(0);  // Set to "red".
    }
  }
}


void create_street_network() {
  int zero = 0;
  hipMemcpyToSymbol(HIP_SYMBOL(dev_num_cells), &zero, sizeof(int), 0,
                     hipMemcpyHostToDevice);
  hipMalloc(&h_nodes, sizeof(Node)*kNumIntersections);
  hipMemcpyToSymbol(HIP_SYMBOL(d_nodes), &h_nodes, sizeof(Node*), 0,
                     hipMemcpyHostToDevice);
  hipMalloc(&h_traffic_lights, sizeof(TrafficLight)*kNumIntersections);
  hipMemcpyToSymbol(HIP_SYMBOL(d_traffic_lights), &h_traffic_lights,
                     sizeof(TrafficLight*), 0, hipMemcpyHostToDevice);
  gpuErrchk(hipDeviceSynchronize());

  kernel_create_nodes<<<
      (kNumIntersections + kNumBlockSize - 1) / kNumBlockSize,
      kNumBlockSize>>>();
  gpuErrchk(hipDeviceSynchronize());

  kernel_create_edges<<<
      (kNumIntersections + kNumBlockSize - 1) / kNumBlockSize,
      kNumBlockSize>>>();
  gpuErrchk(hipDeviceSynchronize());

  kernel_create_traffic_lights<<<
      (kNumIntersections + kNumBlockSize - 1) / kNumBlockSize,
      kNumBlockSize>>>();
  gpuErrchk(hipDeviceSynchronize());

  // Allocate helper data structures for rendering.
  hipMemcpyFromSymbol(&host_num_cells, HIP_SYMBOL(dev_num_cells), sizeof(int), 0,
                       hipMemcpyDeviceToHost);
  hipMalloc(&host_Cell_pos_x, sizeof(float)*host_num_cells);
  hipMemcpyToSymbol(HIP_SYMBOL(dev_Cell_pos_x), &host_Cell_pos_x, sizeof(float*), 0,
                     hipMemcpyHostToDevice);
  hipMalloc(&host_Cell_pos_y, sizeof(float)*host_num_cells);
  hipMemcpyToSymbol(HIP_SYMBOL(dev_Cell_pos_y), &host_Cell_pos_y, sizeof(float*), 0,
                     hipMemcpyHostToDevice);
  hipMalloc(&host_Cell_occupied, sizeof(bool)*host_num_cells);
  hipMemcpyToSymbol(HIP_SYMBOL(dev_Cell_occupied), &host_Cell_occupied, sizeof(bool*), 0,
                     hipMemcpyHostToDevice);
  host_data_Cell_pos_x = (float*) malloc(sizeof(float)*host_num_cells);
  host_data_Cell_pos_y = (float*) malloc(sizeof(float)*host_num_cells);
  host_data_Cell_occupied = (bool*) malloc(sizeof(bool)*host_num_cells);

  printf("Number of cells: %i\n", host_num_cells);
}


void step_traffic_lights() { 
  // TODO: Consider migrating this to SoaAlloc.
  kernel_traffic_light_step<<<
      (kNumIntersections + kNumBlockSize - 1) / kNumBlockSize,
      kNumBlockSize>>>();
  gpuErrchk(hipDeviceSynchronize());
}


void transfer_data() {
  int zero = 0;
  hipMemcpyToSymbol(HIP_SYMBOL(dev_num_cells), &zero, sizeof(int), 0,
                     hipMemcpyHostToDevice);

  allocator_handle->parallel_do<Cell, &Cell::add_to_rendering_array>();

  hipMemcpy(host_data_Cell_pos_x, host_Cell_pos_x,
             sizeof(float)*host_num_cells, hipMemcpyDeviceToHost);
  hipMemcpy(host_data_Cell_pos_y, host_Cell_pos_y,
             sizeof(float)*host_num_cells, hipMemcpyDeviceToHost);
  hipMemcpy(host_data_Cell_occupied, host_Cell_occupied,
             sizeof(float)*host_num_cells, hipMemcpyDeviceToHost);

  gpuErrchk(hipDeviceSynchronize());
}


void step() {
  allocator_handle->parallel_do<ProducerCell, &ProducerCell::create_car>();
  
  step_traffic_lights();
  allocator_handle->parallel_do<Car, &Car::step_prepare_path>();
  allocator_handle->parallel_do<Car, &Car::step_move>();
}



int main(int /*argc*/, char** /*argv*/) {
  if (kOptionRender) {
    init_renderer();
  }

  // Create new allocator.
  allocator_handle = new AllocatorHandle<AllocatorT>();
  AllocatorT* dev_ptr = allocator_handle->device_pointer();
  hipMemcpyToSymbol(HIP_SYMBOL(device_allocator), &dev_ptr, sizeof(AllocatorT*), 0,
                     hipMemcpyHostToDevice);

  create_street_network();

  for (int i = 0; i < kNumIterations; ++i) {
    if (kOptionRender) {
      transfer_data();
      draw(host_data_Cell_pos_x, host_data_Cell_pos_y, host_data_Cell_occupied,
           host_num_cells);
    }

    step();
  }

  if (kOptionRender) {
    close_renderer();
  }
}
