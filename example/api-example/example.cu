#include "hip/hip_runtime.h"
#include "allocator/soa_allocator.h"
#include "allocator/soa_base.h"
#include "allocator/allocator_handle.h"

// Pre-declare all classes.
class Foo;
class Bar;

// Declare allocator type. First argument is max. number of objects that can be created.
using AllocatorT = SoaAllocator<64*64*64*64, Foo, Bar>;

// Allocator handles.
__device__ AllocatorT* device_allocator;
AllocatorHandle<AllocatorT>* allocator_handle;

class Foo : public SoaBase<AllocatorT> {
 public:
  // Pre-declare types of all fields.
  using FieldTypes = std::tuple<float, int, char>;
  
  // Declare fields.
  SoaField<Foo, 0> field1_;  // float
  SoaField<Foo, 1> field2_;  // int
  SoaField<Foo, 2> field3_;  // char
  
  __device__ Foo(float f1, int f2, char f3) : field1_(f1), field2_(f2), field3_(f3) {}
 
  __device__ void qux() {
    field1_ = field2_ + field3_;
  }
};

class Bar : public SoaBase<AllocatorT> {
 public:
  using FieldTypes = std::tuple<int, int, int>;
};

__global__ void create_objects() {
  device_allocator->make_new<Foo>(1.0f, threadIdx.x, 2);
  // Delete objects with: device_allocator->free<Foo>(ptr)
}

int main(int argc, char** argv) {
  // Optional, for debugging.
  AllocatorT::DBG_print_stats();
  
  // Create new allocator.
  allocator_handle = new AllocatorHandle<AllocatorT>();
  AllocatorT* dev_ptr = allocator_handle->device_pointer();
  hipMemcpyToSymbol(HIP_SYMBOL(device_allocator), &dev_ptr, sizeof(AllocatorT*), 0,
                     hipMemcpyHostToDevice);

  // Create 2048 objects.
  create_objects<<<32, 64>>>();
  hipDeviceSynchronize();

  // Call Foo::qux on all 2048 objects.
  allocator_handle->parallel_do<Foo, &Foo::qux>();
}
