#include "hip/hip_runtime.h"
#include <chrono>
#include <stdio.h>

#include "example/game-of-life/soa/configuration.h"
#include "example/game-of-life/soa/rendering.h"


using CellT = char;


#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}


// Data structure.
__device__ CellT* dev_cells;
__device__ CellT* dev_next_cells;
CellT* host_cells;
CellT* host_next_cells;

// Only for rendering.
CellT host_render_cells[SIZE_X*SIZE_Y];


__global__ void initialize_cells() {
  for (int i = threadIdx.x + blockDim.x * blockIdx.x;
       i < SIZE_X*SIZE_Y; i += blockDim.x * gridDim.x) {
    dev_cells[i] = 0;
    dev_next_cells[i] = 0;
  }
}


__global__ void load_game(int* cell_ids, int num_cells) {
  for (int i = threadIdx.x + blockDim.x * blockIdx.x;
       i < num_cells; i += blockDim.x * gridDim.x) {
    dev_cells[cell_ids[i]] = 1;
    dev_next_cells[cell_ids[i]] = 1;
  }
}


__global__ void update() {
  for (int i = threadIdx.x + blockDim.x * blockIdx.x;
       i < SIZE_X*SIZE_Y; i += blockDim.x * gridDim.x) {
    // Check all neigboring cells.
    int num_alive = 0;

    int cell_x = i % SIZE_X;
    int cell_y = i / SIZE_X;

    for (int dx = -1; dx < 2; ++dx) {
      for (int dy = -1; dy < 2; ++dy) {
        int nx = cell_x + dx;
        int ny = cell_y + dy;

        if ((dx != 0 || dy != 0)
             && nx > -1 && nx < SIZE_X && ny > -1 && ny < SIZE_Y) {
          num_alive += dev_cells[i];
        }
      }
    }

    if (num_alive < 2 || num_alive > 3) {
      dev_next_cells[i] = 0;
    } else if (num_alive == 3) {
      dev_next_cells[i] = 1;
    }
  }
}


__global__ void swap_arrays() {
  if (threadIdx.x + blockDim.x * blockIdx.x == 0) {
    auto* tmp = dev_cells;
    dev_cells = dev_next_cells;
    dev_next_cells = tmp;
  }
}


int encode_cell_coords(int x, int y) {
  return SIZE_X*y + x;
}


void load_glider() {
  // Create data set.
  int cell_ids[5];
  cell_ids[0] = encode_cell_coords(1, 0);
  cell_ids[1] = encode_cell_coords(2, 1);
  cell_ids[2] = encode_cell_coords(0, 2);
  cell_ids[3] = encode_cell_coords(1, 2);
  cell_ids[4] = encode_cell_coords(2, 2);

  int* dev_cell_ids;
  hipMalloc(&dev_cell_ids, sizeof(int)*5);
  hipMemcpy(dev_cell_ids, cell_ids, sizeof(int)*5, hipMemcpyHostToDevice);

  load_game<<<1, 5>>>(dev_cell_ids, 5);
  gpuErrchk(hipDeviceSynchronize());
  hipFree(dev_cell_ids);
}


void render() {
  hipMemcpy(host_render_cells, host_cells, sizeof(CellT)*SIZE_X*SIZE_Y,
             hipMemcpyDeviceToHost);
  draw(host_render_cells);
}


int main(int argc, char** argv) {
  if (OPTION_DRAW) {
    init_renderer();
  }

  // Allocate device memory.
  hipMalloc(&host_cells, sizeof(CellT)*SIZE_X*SIZE_Y);
  hipMalloc(&host_next_cells, sizeof(CellT)*SIZE_X*SIZE_Y);
  hipMemcpyToSymbol(HIP_SYMBOL(dev_cells), &host_cells, sizeof(CellT*), 0,
                     hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(dev_next_cells), &host_next_cells, sizeof(CellT*), 0,
                     hipMemcpyHostToDevice);

  // Initialize cells.
  initialize_cells<<<128, 128>>>();
  gpuErrchk(hipDeviceSynchronize());

  // Load data set.
  load_glider();

  // Run simulation.
  for (int i = 0; i < 500; ++i) {
    printf("Iteration %i\n", i);
    // TODO: Tune launch configuration.
    update<<<128, 256>>>();
    gpuErrchk(hipDeviceSynchronize());
    swap_arrays<<<1, 1>>>();
    gpuErrchk(hipDeviceSynchronize());

    if (OPTION_DRAW) {
      render();
    }
  }

  if (OPTION_DRAW) {
    close_renderer();
  }

  // Free device memory.
  hipFree(host_cells);
  hipFree(host_next_cells);

  return 0;
}
